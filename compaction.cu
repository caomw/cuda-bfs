#include "hip/hip_runtime.h"
#include "compaction_kernels.cu"

inline
bool isPowerOfTwo(int n) {
    return (n & (n-1)) == 0;
}

inline
int floorPow2(int n) {
    int exp;
    frexp((float)n, &exp);
    return 1 << (exp - 1);
}

#define BLOCK_SIZE 256

unsigned **scanBlockSums;
unsigned int numEltsAllocated = 0;
unsigned int numLevelsAllocated = 0;

__host__
void preallocBlockSums(unsigned int maxNumElements) {
    numEltsAllocated = maxNumElements;

    unsigned int blockSize = BLOCK_SIZE;
    unsigned int numElts = maxNumElements;

    int level = 0;

    do {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) {
            level++;
        }
        numElts = numBlocks;
    } while (numElts > 1);

    scanBlockSums = (unsigned**) malloc(level * sizeof(unsigned*));
    numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) {
            gpuErrchk(hipMalloc(&scanBlockSums[level++], numBlocks * sizeof(unsigned)));
        }
        numElts = numBlocks;
    } while (numElts > 1);
}

__host__
void deallocBlockSums() {
    for (unsigned int i = 0; i < numLevelsAllocated; i++) {
        hipFree(scanBlockSums[i]);
    }
    
    free(scanBlockSums);

    scanBlockSums = 0;
    numEltsAllocated = 0;
    numLevelsAllocated = 0;
}

__host__
void prescanArrayRecursive(unsigned *outArray, 
                           const unsigned *inArray, 
                           int numElements, 
                           int level) {

    unsigned int blockSize = BLOCK_SIZE;
    unsigned int numBlocks = 
        max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    unsigned int numEltsLastBlock = 
        numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);    
        
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = 
            sizeof(unsigned) * (2 * numThreadsLastBlock + extraSpace);
    }

    // Avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = 
        sizeof(unsigned) * (numEltsPerBlock + extraSpace);

    dim3 grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3 threads(numThreads, 1, 1);

    // Main action

    if (numBlocks > 1) {
        prescan<true, false> <<< grid, threads, sharedMemSize >>> (
            outArray, inArray, scanBlockSums[level], numThreads * 2, 0, 0);
        
        if (np2LastBlock) {
            prescan<true, true> <<< 1, numThreadsLastBlock, sharedMemLastBlock >>> (
                outArray, inArray, scanBlockSums[level], numEltsLastBlock, 
                numBlocks - 1, numElements - numEltsLastBlock);
        }

        prescanArrayRecursive(scanBlockSums[level], scanBlockSums[level], numBlocks, level+1);

        uniformAdd <<< grid, threads >>> (
            outArray, scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);

        if (np2LastBlock) {
            uniformAdd <<<1, numThreadsLastBlock>>> (
                outArray, scanBlockSums[level], numEltsLastBlock, 
                numBlocks - 1, numElements - numEltsLastBlock);
        }
    } else if (isPowerOfTwo(numElements)) {
        prescan<false, false> <<<grid, threads, sharedMemSize>>> (
            outArray, inArray, 0, numThreads * 2, 0, 0);
    } else {
         prescan<false, true> <<<grid, threads, sharedMemSize>>> (
            outArray, inArray, 0, numElements, 0, 0);
    }
}

__host__
void prescanArray(unsigned *outArray, unsigned *inArray, int numElements) {
    prescanArrayRecursive(outArray, inArray, numElements, 0);
}
