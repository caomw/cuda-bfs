#include "bfs.hpp"
#include "kernels.cuh"

__device__
void setInt(unsigned *ptr, unsigned value) {
    *ptr = value;
}

__host__
void BFS(Graph & graph, unsigned sourceVertex, std::vector<unsigned> & distances) {

    assert(sizeof(unsigned) == 4);
    
    distances.clear();
    distances.resize(graph.size());

    // Convert the graph to GPU representation

    size_t totalEdges = 0;
    for (auto & neighborsList : graph) {
        totalEdges += neighborsList.size();
    }

    std::vector<unsigned> V(graph.size());
    std::vector<unsigned> E;
    E.reserve(totalEdges);

    for (size_t v = 0; v < graph.size(); ++v) {
        V[v] = E.size();
        for (int neighbor : graph[v]) {
            E.push_back(neighbor);
        }
    }

    // Memory allocation and setup

    unsigned *d_V, *d_E;
    unsigned *d_F, *d_X, *d_C, *d_Fu;
    
    size_t memSize = graph.size() * sizeof(unsigned);
    
    gpuErrchk(hipMalloc(&d_F, memSize));
    gpuErrchk(hipMemset(d_F, 0, memSize));
    setInt(d_F + sourceVertex, 1); // add source to frontier

    gpuErrchk(hipMalloc(&d_X, memSize));
    gpuErrchk(hipMemset(d_X, 0, memSize));
    setInt(d_X + sourceVertex, 1); // set source as visited

    gpuErrchk(hipMalloc(&d_C, memSize));
    gpuErrchk(hipMemset(d_X, 0, memSize));
    setInt(d_C + sourceVertex, 0); // set zero distance to source

    gpuErrchk(hipMalloc(&d_Fu, memSize));

    gpuErrchk(hipMalloc(&d_V, memSize));
    gpuErrchk(hipMemcpy(d_V, V.data(), memSize, hipMemcpyHostToDevice));

    size_t memSizeE = totalEdges * sizeof(unsigned);
    gpuErrchk(hipMalloc(&d_E, memSizeE));
    gpuErrchk(hipMemcpy(d_E, E.data(), memSizeE, hipMemcpyHostToDevice));

    bool terminate = false;

    while (!terminate) {
        // launch kernel 1
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // launch kernel 2
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        terminate = true;
        // copy terminate from GPU
    }

    gpuErrchk(hipMemcpy(distances.data(), d_C, memSize, hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(d_F));
    gpuErrchk(hipFree(d_X));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_Fu));
    gpuErrchk(hipFree(d_V));
    gpuErrchk(hipFree(d_E));
}
