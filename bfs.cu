#include "hip/hip_runtime.h"
#include "bfs.hpp"
#include "bfs_kernels.cuh"
#include "compaction.cuh"
#include <stdio.h>

extern __managed__ unsigned terminate;
extern __managed__ unsigned numActiveThreads;

__host__
void setUInt(unsigned *address, unsigned value) {
    gpuErrchk(hipMemcpy(address, &value, sizeof(unsigned), hipMemcpyHostToDevice));
}

__global__
void output(int N, unsigned *ptr) {
    printf("Prefix sums: ");
    for (int i = 0; i < N; ++i) {
        printf("%u ", ptr[i]);
    }
    printf("\n");
}

__host__
void BFS(Graph & graph, unsigned sourceVertex, std::vector<unsigned> & distances) {

    assert(sizeof(unsigned) == 4);
    
    distances.clear();
    distances.resize(graph.size());

    // Convert the graph to GPU representation

    size_t totalEdges = 0;
    for (auto & neighborsList : graph) {
        totalEdges += neighborsList.size();
    }

    std::vector<unsigned> V(graph.size() + 1);
    std::vector<unsigned> E;
    E.reserve(totalEdges);

    for (size_t v = 0; v < graph.size(); ++v) {
        V[v] = E.size();
        for (int neighbor : graph[v]) {
            E.push_back(neighbor);
        }
    }
    V[graph.size()] = totalEdges;

printf("V: ");
for (int x : V) printf("%u ", x);
printf("\n");

printf("E: ");
for (int x : E) printf("%u ", x);
printf("\n");

    // Memory allocation and setup

    unsigned *d_V, *d_E;
    unsigned *d_F, *d_X, *d_C, *d_Fu;
    unsigned *activeMask, *prefixSums;

    size_t memSize = (graph.size() + 1) * sizeof(unsigned);
    
    gpuErrchk(hipMalloc(&d_F, memSize));
    gpuErrchk(hipMemset(d_F, FALSE, memSize));
    setUInt(d_F + sourceVertex, TRUE); // add source to frontier

    gpuErrchk(hipMalloc(&d_X, memSize));
    gpuErrchk(hipMemset(d_X, FALSE, memSize));
    setUInt(d_X + sourceVertex, TRUE); // set source as visited

    gpuErrchk(hipMalloc(&d_C, memSize));
    gpuErrchk(hipMemset(d_C, 255, memSize));
    setUInt(d_C + sourceVertex, FALSE); // set zero distance to source

    gpuErrchk(hipMalloc(&d_Fu, memSize));

    gpuErrchk(hipMalloc(&d_V, memSize));
    gpuErrchk(hipMemcpy(d_V, V.data(), memSize, hipMemcpyHostToDevice));

    size_t memSizeE = totalEdges * sizeof(unsigned);
    gpuErrchk(hipMalloc(&d_E, memSizeE));
    gpuErrchk(hipMemcpy(d_E, E.data(), memSizeE, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&activeMask, memSize));
    setUInt(activeMask + 0, sourceVertex); // set thread #source as active
    numActiveThreads = 1;

    gpuErrchk(hipMalloc(&prefixSums, memSize));
    preallocBlockSums(graph.size() + 1);

    // Main loop

    printf("Settled\n"); fflush(stdout);

    const size_t prefixSumGridSize = 
        (graph.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;

    while (true) {

        terminate = TRUE;

        const size_t gridSize = 
            (numActiveThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;

        printf("Kernel 1, <<<%d, %d>>>\n", gridSize, BLOCK_SIZE); fflush(stdout);
        // launch kernel 1
        BFSKernel1 <<<gridSize, BLOCK_SIZE>>> (graph.size(), d_V, d_E, d_F, d_X, d_C, d_Fu);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        printf("Kernel 2, <<<%d, %d>>>...", gridSize, BLOCK_SIZE); fflush(stdout);
        // launch kernel 2
        BFSKernel2 <<<gridSize, BLOCK_SIZE>>> (graph.size(), d_F, d_X, d_Fu);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        printf("done\n"); fflush(stdout);

        if (terminate) {
            break;
        } else {
            // Get prefix sums of F
            prescanArray(prefixSums, d_F, graph.size() + 1);
            
            output <<<1,1>>> (graph.size(), prefixSums);
            gpuErrchk(hipDeviceSynchronize());

            const size_t gridSizeCompaction = (graph.size() + BLOCK_SIZE - 1) / BLOCK_SIZE;
            compactSIMD <<<gridSizeCompaction, BLOCK_SIZE>>> (prefixSums, activeMask);

            printf("Kernel 3, <<<1, 1>>>\n"); fflush(stdout);
            getActiveMaskTemp <<<1, 1>>> (graph.size(), d_F, activeMask);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }
    }

    // Download result

    gpuErrchk(hipMemcpy(distances.data(), d_C, memSize-sizeof(unsigned), hipMemcpyDeviceToHost));

    // Free memory

    gpuErrchk(hipFree(d_F));
    gpuErrchk(hipFree(d_X));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_Fu));
    gpuErrchk(hipFree(d_V));
    gpuErrchk(hipFree(d_E));
    gpuErrchk(hipFree(activeMask));
    deallocBlockSums();
    gpuErrchk(hipFree(prefixSums));
}
