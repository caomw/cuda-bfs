#include "bfs.hpp"
#include "kernels.cuh"

__host__
void setUInt(unsigned *address, unsigned value) {

}

__host__
void BFS(Graph & graph, unsigned sourceVertex, std::vector<unsigned> & distances) {

    assert(sizeof(unsigned) == 4);
    
    distances.clear();
    distances.resize(graph.size());

    // Convert the graph to GPU representation

    size_t totalEdges = 0;
    for (auto & neighborsList : graph) {
        totalEdges += neighborsList.size();
    }

    std::vector<unsigned> V(graph.size() + 1);
    std::vector<unsigned> E;
    E.reserve(totalEdges);

    for (size_t v = 0; v < graph.size(); ++v) {
        V[v] = E.size();
        for (int neighbor : graph[v]) {
            E.push_back(neighbor);
        }
    }
    V[graph.size()] = totalEdges;

    // Memory allocation and setup

    unsigned *d_V, *d_E;
    unsigned *d_F, *d_X, *d_C, *d_Fu;

    size_t memSize = (graph.size() + 1) * sizeof(unsigned);
    
    gpuErrchk(hipMalloc(&d_F, memSize));
    gpuErrchk(hipMemset(d_F, FALSE, memSize));
    setUInt(d_F + sourceVertex, TRUE); // add source to frontier

    gpuErrchk(hipMalloc(&d_X, memSize));
    gpuErrchk(hipMemset(d_X, FALSE, memSize));
    setUInt(d_X + sourceVertex, TRUE); // set source as visited

    gpuErrchk(hipMalloc(&d_C, memSize));
    setUInt(d_C + sourceVertex, FALSE); // set zero distance to source

    gpuErrchk(hipMalloc(&d_Fu, memSize));

    gpuErrchk(hipMalloc(&d_V, memSize));
    gpuErrchk(hipMemcpy(d_V, V.data(), memSize, hipMemcpyHostToDevice));

    size_t memSizeE = totalEdges * sizeof(unsigned);
    gpuErrchk(hipMalloc(&d_E, memSizeE));
    gpuErrchk(hipMemcpy(d_E, E.data(), memSizeE, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&activeMask, memSize));
    gpuErrchk(hipMemset(activeMask, FALSE, memSize));
    setUInt(activeMask + sourceVertex, TRUE); // set thread #source as active
    numActiveThreads = 1;

    terminate = TRUE;

    // Main loop

    const size_t prefixSumGridSize = 
        (graph.size() + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    while (true) {
        const size_t gridSize = 
            (numActiveThreads + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

        // launch kernel 1
        BFSKernel1 <<<gridSize, MAX_THREADS_PER_BLOCK>>> (graph.size(), d_V, d_E, d_F, d_X, d_C, d_Fu);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // launch kernel 2
        BFSKernel2 <<<gridSize, MAX_THREADS_PER_BLOCK>>> (graph.size(), d_F, d_X, d_Fu);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // copy terminate from GPU

        if (terminate) {
            break;
        } else {
            // Get active threads list
            //prefixSum <<<prefixSumGridSize, MAX_THREADS_PER_BLOCK>>> (d_F, activeMask);
            //gather <<<
            getActiveMaskTemp <<<1, 1>>> (d_F, activeMask);

            //numActiveThreads
        }
    }

    // Download result

    gpuErrchk(hipMemcpy(distances.data(), d_C, memSize, hipMemcpyDeviceToHost));

    // Free memory

    gpuErrchk(hipFree(d_F));
    gpuErrchk(hipFree(d_X));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_Fu));
    gpuErrchk(hipFree(d_V));
    gpuErrchk(hipFree(d_E));
    gpuErrchk(hipFree(activeMask));
}
